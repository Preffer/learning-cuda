#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <boost/timer.hpp>
#include <boost/format.hpp>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace boost;

__global__ void multiplyKernel(const float* A, const float* B, float* C, const unsigned int R);
void logTime(const string& message);

int main(int argc, char* argv[]) {
	if (argc != 2 && argc != 3) {
		cout << format("Usage: %1% <R> <save?>") % argv[0] << endl;
		return EXIT_FAILURE;
	}

	logTime("Launched");
	const int R = stoi(argv[1]);

	float* A = new float[R * R];
	float* B = new float[R * R];
	float* C = new float[R * R];
	logTime("Host memory allocated");

	for (int i = 0; i < R * R; i++) {
		A[i] = rand();
		B[i] = rand();
	}
	logTime("Matrix randomly filled");

	try {
		float* devA = NULL;
		float* devB = NULL;
		float* devC = NULL;
		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) throw runtime_error("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");

		cudaStatus = hipMalloc((void**)&devA, R * R * sizeof(float));
		if (cudaStatus != hipSuccess) throw runtime_error("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&devB, R * R * sizeof(float));
		if (cudaStatus != hipSuccess) throw runtime_error("hipMalloc failed!");

		cudaStatus = hipMalloc((void**)&devC, R * R * sizeof(float));
		if (cudaStatus != hipSuccess) throw runtime_error("hipMalloc failed!");

		cudaStatus = hipMemcpy(devA, A, R * R * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) throw runtime_error("hipMemcpy failed!");

		cudaStatus = hipMemcpy(devB, B, R * R * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) throw runtime_error("hipMemcpy failed!");

		logTime("Ready to launch kernel");
		timer t;
		multiplyKernel << <R, R >> >(devA, devB, devC, R);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) throw runtime_error((format("addKernel launch failed: %1%") % (hipGetErrorString(cudaStatus))).str());

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) throw runtime_error((format("hipDeviceSynchronize returned error code %1% after launching addKernel!") % cudaStatus).str());

		logTime("Kernel finish");
		double gflops = pow(R, 3) / 536870912 / t.elapsed();
		cout << format("%1% GFLOPS / %2%s Computing Time") % gflops % t.elapsed() << endl;

		cudaStatus = hipMemcpy(C, devC, R * R * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) throw runtime_error("hipMemcpy failed!");

		hipFree(devA);
		hipFree(devB);
		hipFree(devC);

		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) throw runtime_error("hipDeviceReset failed!");

	} catch (const runtime_error& e) {
		cerr << e.what() << endl;
		return EXIT_FAILURE;
	}

	logTime("Back to host");
	cout << format("Result Matrix Pointer: %1$p") % C << endl;

	if (argc == 3) {
		ofstream outA("A.csv");
		ofstream outB("B.csv");
		ofstream outC("C.csv");
		for (int row = 0; row < R; row++) {
			for (int col = 0; col < R; col++) {
				outA << A[row * R + col] << " ";
				outB << B[row * R + col] << " ";
				outC << C[row * R + col] << " ";
			}
			outA << endl;
			outB << endl;
			outC << endl;
		}
		logTime("Save finish");
	}

	delete[] A;
	delete[] B;
	delete[] C;

	logTime("Finish");
	return EXIT_SUCCESS;
}

__global__ void multiplyKernel(const float* A, const float* B, float* C, const unsigned int R) {
	int row = blockIdx.x;
	int col = threadIdx.x;
	float sum = 0;

	for (int i = 0; i < R; i++) {
		sum += A[row * R + i] * B[i * R + col];
	}

	C[row * R + col] = sum;
}

void logTime(const string& message) {
	static timer t;
	cout << boost::format("[%1$.3f] %2%") % t.elapsed() % message << endl;
}
